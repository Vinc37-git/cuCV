/**
 * @file cumat.cu
 * @author Vincent Hackstein (vinc37-git)
 * @brief 
 * @version 0.1
 * @date 2022-05-25
 * 
 * @copyright Copyright (c) 2022
 * 
 */


#include "cumat.h"


template <typename T>
cuCV::CuMat<T>::CuMat() { }


template <typename T>
cuCV::CuMat<T>::CuMat(Mat<T> & mat) 
        : Mat<T>(mat.getWidth(), mat.getHeight(), mat.getNChannels(), NULL, false) {  
    ///< We instanciate Base Class Object "Mat" using parameters of mat. However, dont point to the same data.
    //allocateLike(mat);
}


template <typename T>
cuCV::CuMat<T>::CuMat(int width, int height, int channels) 
        : Mat<T>(width, height, channels, NULL, false) { }


template <typename T>
cuCV::CuMat<T>::CuMat(const CuMat & cuMat)
        : Mat<T>(cuMat.mWidth, cuMat.mHeight, cuMat.mChannels, NULL, false) {
    if (cuMat.mData != NULL) {
        allocateLike(cuMat);
        gpuErrchk(hipMemcpy(this->mData, cuMat.mData, sizeof(T) * cuMat.mWidth * cuMat.mHeight * cuMat.mChannels, hipMemcpyDeviceToDevice)); 
        CUCV_DEBUG_PRINT("Copy: %p copied to %p.", cuMat.mData, this->mData);
    }
}


template <typename T>
cuCV::CuMat<T>::CuMat(CuMat && cuMat)
        : Mat<T>(cuMat.mWidth, cuMat.mHeight, cuMat.mChannels, cuMat.mData, cuMat.mBorrowed) {
    cuMat.mData = NULL;
    CUCV_DEBUG_PRINT("Move: %p swaped with %p.", cuMat.mData, this->mData);
}


template <typename T>
cuCV::CuMat<T>::~CuMat() {
    if (this->mData != NULL && !this->mBorrowed) {
        CUCV_DEBUG_PRINT("%p destroyed.", this->mData);
        gpuErrchk(hipFree(this->mData));
        this->mData = NULL;
    }
}


template <typename T>
cuCV::CuMat<T> & cuCV::CuMat<T>::operator=(CuMat cuMat) {
    std::swap(this->mData, cuMat.mData);
    this->cuType = cuMat.cuType;
    this->mWidth = cuMat.mWidth; 
    this->mHeight = cuMat.mHeight; 
    this->mStrideX = cuMat.mStrideX;
    this->mStrideY = cuMat.mStrideY;
    this->mChannels = cuMat.mChannels;
    this->mBorrowed = cuMat.mBorrowed;
    CUCV_DEBUG_PRINT("%p swaped with %p.", cuMat.getDataPtr(), this->mData);
    return * this;
}


template <typename T>
cuCV::CuMat<T> & cuCV::CuMat<T>::operator+=(const CuMat & cuMat) {
    if (this->mData == NULL || cuMat.mData == NULL)
        throw cuCV::exception::NullPointer("Calculation failed. One or more operands point to NULL data!");

    if (!compareDim(* this, cuMat))
        throw cuCV::exception::DimensionMismatch(* this, cuMat);

    // Construct Grid. As for images usually cols && rows >> nCh we do not launch a whole thread-block in z dimension.
    const dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 blocks((this->mWidth + threads.x - 1) / threads.x, (this->mHeight + threads.y - 1) / threads.y, this->mChannels);

    /// Perform Math
    cuCV::kernel::add<<<blocks, threads>>>(this->kernel(), this->kernel(), cuMat.kernel());

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    return * this;    
}


template <typename T>
cuCV::CuMat<T> & cuCV::CuMat<T>::operator+=(T alpha) {
    if (this->mData == NULL)
        throw cuCV::exception::NullPointer("Calculation failed. One or more operands point to NULL data!");

    // Construct Grid. As for images usually cols && rows >> nCh we do not launch a whole thread-block in z dimension.
    const dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 blocks((this->mWidth + threads.x - 1) / threads.x, (this->mHeight + threads.y - 1) / threads.y, this->mChannels);

    /// Perform Math
    cuCV::kernel::add<<<blocks, threads>>>(this->kernel(), this->kernel(), alpha);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    return * this;    
}


template <typename T>
cuCV::CuMat<T> cuCV::CuMat<T>::operator+(const CuMat & cuMat) const {
    cuCV::CuMat<T> sum(*this);  // Copy `this`
    sum += cuMat;
    return sum;
}


template <typename T>
cuCV::CuMat<T> cuCV::CuMat<T>::operator+(const T alpha) const {
    cuCV::CuMat<T> sum(*this);  // Copy `this`
    sum += alpha;
    return sum;
}



template <typename T>
cuCV::CuMat<T> & cuCV::CuMat<T>::operator-=(const CuMat & cuMat) {
    if (this->mData == NULL || cuMat.mData == NULL)
        throw cuCV::exception::NullPointer("Calculation failed. One or more operands point to NULL data!");

    if (!compareDim(* this, cuMat))
        throw cuCV::exception::DimensionMismatch(* this, cuMat);

    // Construct Grid. As for images usually cols && rows >> nCh we do not launch a whole thread-block in z dimension.
    const dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 blocks((this->mWidth + threads.x - 1) / threads.x, (this->mHeight + threads.y - 1) / threads.y, this->mChannels);

    /// Perform Math
    cuCV::kernel::dif<<<blocks, threads>>>(this->kernel(), this->kernel(), cuMat.kernel());

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    return * this;    
}


template <typename T>
cuCV::CuMat<T> & cuCV::CuMat<T>::operator-=(T alpha) {
    if (this->mData == NULL)
        throw cuCV::exception::NullPointer("Calculation failed. One or more operands point to NULL data!");

    // Construct Grid. As for images usually cols && rows >> nCh we do not launch a whole thread-block in z dimension.
    const dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 blocks((this->mWidth + threads.x - 1) / threads.x, (this->mHeight + threads.y - 1) / threads.y, this->mChannels);

    /// Perform Math
    cuCV::kernel::dif<<<blocks, threads>>>(this->kernel(), this->kernel(), alpha);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    return * this;    
}


template <typename T>
cuCV::CuMat<T> cuCV::CuMat<T>::operator-(const CuMat & cuMat) const {
    cuCV::CuMat<T> sum(*this);  // Copy `this`
    sum -= cuMat;
    return sum;
}


template <typename T>
cuCV::CuMat<T> cuCV::CuMat<T>::operator-(const T alpha) const {
    cuCV::CuMat<T> sum(*this);  // Copy `this`
    sum -= alpha;
    return sum;
}


template <typename T>
cuCV::CuMat<T> & cuCV::CuMat<T>::operator*=(const CuMat & cuMat) {
    if (this->mData == NULL || cuMat.mData == NULL)
        throw cuCV::exception::NullPointer("Calculation failed. One or more operands point to NULL data!");

    if (!compareDim(* this, cuMat))
        throw cuCV::exception::DimensionMismatch(* this, cuMat);

    // Construct Grid. As for images usually cols && rows >> nCh we do not launch a whole thread-block in z dimension.
    const dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 blocks((this->mWidth + threads.x - 1) / threads.x, (this->mHeight + threads.y - 1) / threads.y, this->mChannels);

    /// Perform Math
    cuCV::kernel::mul<<<blocks, threads>>>(this->kernel(), this->kernel(), cuMat.kernel());

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    return * this;    
}


template <typename T>
cuCV::CuMat<T> & cuCV::CuMat<T>::operator*=(T alpha) {
    if (this->mData == NULL)
        throw cuCV::exception::NullPointer("Calculation failed. One or more operands point to NULL data!");

    // Construct Grid. As for images usually cols && rows >> nCh we do not launch a whole thread-block in z dimension.
    const dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 blocks((this->mWidth + threads.x - 1) / threads.x, (this->mHeight + threads.y - 1) / threads.y, this->mChannels);

    /// Perform Math
    cuCV::kernel::mul<<<blocks, threads>>>(this->kernel(), this->kernel(), alpha);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    return * this;    
}


template <typename T>
cuCV::CuMat<T> cuCV::CuMat<T>::operator*(const CuMat & cuMat) const {
    cuCV::CuMat<T> sum(*this);  // Copy `this`
    sum *= cuMat;
    return sum;
}


template <typename T>
cuCV::CuMat<T> cuCV::CuMat<T>::operator*(const T alpha) const {
    cuCV::CuMat<T> sum(*this);  // Copy `this`
    sum *= alpha;
    return sum;
}


template <typename T>
cuCV::CuMat<T> & cuCV::CuMat<T>::operator/=(const CuMat & cuMat) {
    if (this->mData == NULL || cuMat.mData == NULL)
        throw cuCV::exception::NullPointer("Calculation failed. One or more operands point to NULL data!");

    if (!compareDim(* this, cuMat))
        throw cuCV::exception::DimensionMismatch(* this, cuMat);

    // Construct Grid. As for images usually cols && rows >> nCh we do not launch a whole thread-block in z dimension.
    const dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 blocks((this->mWidth + threads.x - 1) / threads.x, (this->mHeight + threads.y - 1) / threads.y, this->mChannels);

    /// Perform Math
    cuCV::kernel::div<<<blocks, threads>>>(this->kernel(), this->kernel(), cuMat.kernel());

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    return * this;    
}


template <typename T>
cuCV::CuMat<T> & cuCV::CuMat<T>::operator/=(T alpha) {
    if (this->mData == NULL)
        throw cuCV::exception::NullPointer("PointerError: one or more operands point to NULL data!");

    // Construct Grid. As for images usually cols && rows >> nCh we do not launch a whole thread-block in z dimension.
    const dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 blocks((this->mWidth + threads.x - 1) / threads.x, (this->mHeight + threads.y - 1) / threads.y, this->mChannels);

    /// Perform Math
    cuCV::kernel::div<<<blocks, threads>>>(this->kernel(), this->kernel(), alpha);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    return * this;    
}


template <typename T>
cuCV::CuMat<T> cuCV::CuMat<T>::operator/(const CuMat & cuMat) const {
    cuCV::CuMat<T> sum(*this);  // Copy `this`
    sum /= cuMat;
    return sum;
}


template <typename T>
cuCV::CuMat<T> cuCV::CuMat<T>::operator/(const T alpha) const {
    cuCV::CuMat<T> sum(*this);  // Copy `this`
    sum /= alpha;
    return sum;
}


template <typename T>
void cuCV::CuMat<T>::uploadFrom(const Mat<T> & src) {
    /** Workflow
     * 2. Check dimensions. If uniintialised on device, initialise. If mismatch, throw error.
     * 3. Allocate MEM for device matrix. Dimensions checks makes sure count is right.
     * 4. Transfer data.
     */

    // Check if dimenions match. If dimensions are uninitialized assign them.
    if ((this->mWidth == 0) || (this->mHeight == 0) || (this->mChannels == 0)) {
        this->mWidth = src.getWidth();
        this->mHeight = src.getHeight();
        this->mChannels = src.getNChannels();
        this->mStrideX = src.getStrideX();
        this->mStrideY = src.getStrideY();
    }
    else if (!compareDim(src, * this)) {
        throw cuCV::exception::DimensionMismatch(src, * this);
    }

    /// Allocate Memory on device in 'this' points to NULL. If it points to data, it will throw an exception.
    if (this->mData == NULL)
        allocateLike(src);

    /// Send Memory from src to 'this'. 'this' is on device
    gpuErrchk(hipMemcpy(this->mData, src.getDataPtr(), sizeof(T) * src.getWidth() * src.getHeight() * src.getNChannels(), hipMemcpyHostToDevice));   
}


template <typename T>
void cuCV::CuMat<T>::downloadTo(Mat<T> & dst) const {
    /** Workflow
     * 1. Check if CuMat points to data
     * 2. Check dimensions. If uniintialised, initialised. If mismatch, throw error.
     * 3. Allocate MEM for host matrix. Dimensions checks makes sure count is right.
     * 4. Transfer data.
     */

    if (this->mData == NULL)
        throw cuCV::exception::NullPointer("Download failed. mData of source on device is NULL"); ///< @note undefined behaviour! 

    // Check if dimenions match. If dimensions are uninitialized assign them.
    if ((dst.getWidth() == 0) || (dst.getHeight() == 0) || (dst.getNChannels() == 0)) {
        dst.setWidth(this->mWidth);
        dst.setHeight(this->mHeight);
        dst.setNChannels(this->mChannels);
        dst.setStrideX(this->mStrideX);
        dst.setStrideY(this->mStrideY);
    }
    else if (!compareDim(* this, dst))
        throw cuCV::exception::DimensionMismatch(* this, dst);
    

    // If MEM for destination on host is not allocated yet, allocate.
    if (dst.getDataPtr() == NULL)
        dst.alloc();  // The dim check makes sure the allocated size is always the right one for `this`.
    
    gpuErrchk(hipMemcpy(dst.getDataPtr(), this->mData, sizeof(T) * dst.getWidth() * dst.getHeight() * dst.getNChannels(), hipMemcpyDeviceToHost));
}


template <typename T>
void cuCV::CuMat<T>::allocateLike(const Mat<T> & src) {
    // Check if dimenions match. If dimensions are uninitialized assign them.
    if ((this->mWidth == 0) || (this->mHeight == 0) || (this->mChannels == 0)) {
        this->mWidth = src.getWidth();
        this->mHeight = src.getHeight();
        this->mChannels = src.getNChannels();
        this->mStrideX = src.getStrideX();
        this->mStrideY = src.getStrideY();
    }
    else if (!compareDim(src, * this))
        throw cuCV::exception::DimensionMismatch(src, * this, "allocation");

    /// Allocate Memory
    allocateOnDevice();

    if (this->mData == NULL)
        throw std::bad_alloc(); //("Allocation of VRAM failed.")
}


template <typename T>
void cuCV::CuMat<T>::allocateOnDevice() {
    if (this->mData != NULL) {
        fprintf(stderr, "Warning: Data on device is freed automatically when function allocateOnDevice() is called. Make sure to clear data yourself.");
        clearOnDevice();
    }
    if ((this->mWidth == 0) || (this->mHeight == 0) || (this->mChannels == 0))
        fprintf(stderr, "Warning: Size of CuMat is zero. Allocation has no effect.");

    gpuErrchk(hipMalloc((void**) & this->mData, sizeof(T) * this->mWidth * this->mHeight * this->mChannels));

    CUCV_DEBUG_PRINT("Allocated %ld bytes at %p.", this->getSize() * sizeof(T), this->mData);
    
    if (this->mData == NULL)
        throw std::bad_alloc(); //("Allocation of VRAM failed.")    
}


template <typename T>
void cuCV::CuMat<T>::clearOnDevice() {
    if (this->mData != NULL && !this->mBorrowed) {
        CUCV_DEBUG_PRINT("%p hipFree'd.", this->mData);
        hipFree(this->mData);
        this->mData = NULL;
    }
}


template <typename T>
cuCV::DeviceCuMat<T> cuCV::CuMat<T>::kernel() const {
    return cuCV::DeviceCuMat<T>(* this);
}


template <typename T>
bool cuCV::CuMat<T>::compareDim(const CuMat & A, const CuMat & B) const {
    if (A.getWidth() != B.getWidth() || A.getHeight() != B.getHeight() || A.getNChannels() != B.getNChannels())
        return 0;
    return 1;
}

template <typename T>
bool cuCV::CuMat<T>::compareDim(const CuMat & A, const cuCV::Mat<T> & B) const {
    if (A.getWidth() != B.getWidth() || A.getHeight() != B.getHeight() || A.getNChannels() != B.getNChannels())
        return 0;
    return 1;
}

template <typename T>
bool cuCV::CuMat<T>::compareDim(const cuCV::Mat<T> & A, const CuMat & B) const {
    if (A.getWidth() != B.getWidth() || A.getHeight() != B.getHeight() || A.getNChannels() != B.getNChannels())
        return 0;
    return 1;
}

template <typename T>
bool cuCV::CuMat<T>::compareDim(const cuCV::Mat<T> & A, const cuCV::Mat<T> & B) const {
    if (A.getWidth() != B.getWidth() || A.getHeight() != B.getHeight() || A.getNChannels() != B.getNChannels())
        return 0;
    return 1;
}


template <typename T>
bool cuCV::CuMat<T>::empty() const {
    return this->mData == NULL;
}


/// Explicit template specialization
template class cuCV::CuMat<CUCV_8U>;
template class cuCV::CuMat<CUCV_16U>;
template class cuCV::CuMat<CUCV_32F>;
template class cuCV::CuMat<CUCV_64F>;