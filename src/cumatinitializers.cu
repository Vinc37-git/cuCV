/**
 * @file cumatinitializers.cpp
 * @author Vincent Hackstein (vinc37-git)
 * @brief 
 * @version 0.1
 * @date 2022-08-22
 * 
 * @copyright Copyright (c) 2022
 * 
 */

#include "cumatinitializers.h"


template <typename T>
cuCV::CuMat<T> cuCV::getEmptyOnDevice(int width, int height, int channels) {
    cuCV::CuMat<T> mat(width, height, channels);
    //mat.allocateOnDevice();
    return mat;
}


template <typename T>
cuCV::CuMat<T> cuCV::zerosOnDevice(int width, int height, int channels) {
    cuCV::CuMat<T> mat(width, height, channels);
    mat.allocateOnDevice();

    // Construct Grid. As for images usually cols && rows >> nCh we do not launch a whole thread-block in z dimension.
    const dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 blocks((mat.getWidth() + threads.x - 1) / threads.x, (mat.getHeight() + threads.y - 1) / threads.y, mat.getNChannels());

    /// Perform Math
    cuCV::kernel::zeros<<<blocks, threads>>>(mat.kernel());

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    return mat;
}


template <typename T>
cuCV::CuMat<T> cuCV::onesOnDevice(int width, int height, int channels) {
    cuCV::CuMat<T> mat(width, height, channels);
    mat.allocateOnDevice();

    // Construct Grid. As for images usually cols && rows >> nCh we do not launch a whole thread-block in z dimension.
    const dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 blocks((mat.getWidth() + threads.x - 1) / threads.x, (mat.getHeight() + threads.y - 1) / threads.y, mat.getNChannels());

    /// Perform Math
    cuCV::kernel::ones<<<blocks, threads>>>(mat.kernel());

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    return mat;
}


template <typename T>
cuCV::CuMat<T> cuCV::eyeOnDevice(int width, int height, int channels) {
    cuCV::CuMat<T> mat(width, height, channels);
    mat.allocateOnDevice();

    // Construct Grid. As for images usually cols && rows >> nCh we do not launch a whole thread-block in z dimension.
    const dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 blocks((mat.getWidth() + threads.x - 1) / threads.x, (mat.getHeight() + threads.y - 1) / threads.y, mat.getNChannels());

    /// Perform Math
    cuCV::kernel::eye<<<blocks, threads>>>(mat.kernel());

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    return mat;
}


/// Explicit template specialization
template cuCV::CuMat<CUCV_8U> cuCV::getEmptyOnDevice<CUCV_8U>(const int width, const int height, const int channels);
template cuCV::CuMat<CUCV_16U> cuCV::getEmptyOnDevice<CUCV_16U>(const int width, const int height, const int channels);
template cuCV::CuMat<CUCV_64F> cuCV::getEmptyOnDevice<CUCV_64F>(const int width, const int height, const int channels);

template cuCV::CuMat<CUCV_8U> cuCV::zerosOnDevice<CUCV_8U>(const int width, const int height, const int channels);
template cuCV::CuMat<CUCV_16U> cuCV::zerosOnDevice<CUCV_16U>(const int width, const int height, const int channels);
template cuCV::CuMat<CUCV_64F> cuCV::zerosOnDevice<CUCV_64F>(const int width, const int height, const int channels);

template cuCV::CuMat<CUCV_8U> cuCV::onesOnDevice<CUCV_8U>(const int width, const int height, const int channels);
template cuCV::CuMat<CUCV_16U> cuCV::onesOnDevice<CUCV_16U>(const int width, const int height, const int channels);
template cuCV::CuMat<CUCV_64F> cuCV::onesOnDevice<CUCV_64F>(const int width, const int height, const int channels);

template cuCV::CuMat<CUCV_8U> cuCV::eyeOnDevice<CUCV_8U>(const int width, const int height, const int channels);
template cuCV::CuMat<CUCV_16U> cuCV::eyeOnDevice<CUCV_16U>(const int width, const int height, const int channels);
template cuCV::CuMat<CUCV_64F> cuCV::eyeOnDevice<CUCV_64F>(const int width, const int height, const int channels);