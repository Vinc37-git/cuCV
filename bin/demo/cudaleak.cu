/**
 * 
 * 
 * 
 * 
*/


#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <unistd.h>

#include <hip/hip_runtime.h>

#include "errorhandling.h"

int main(int argc, char ** argv) {
    
    float * data, * devData;
    data = (float *) malloc(1000 * sizeof(float));

    for (int i=0; i<1000; ++i)
        data[i] = 5;

    gpuErrchk(hipMalloc((void**) & devData, sizeof(float) * 1000));
    gpuErrchk(hipMemcpy(devData ,data , sizeof(float) * 1000, hipMemcpyHostToDevice));
    hipFree(devData);
    free(data);

    gpuErrchk(hipDeviceReset());
    return 0;
}